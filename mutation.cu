#include "hip/hip_runtime.h"
/**

  nvcc -arch sm_50 -O3 mutation.cu -o mutation -lcuda -lcufft
 *
 * */

#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void mutate(int *genome,int lengh,hiprandState *state)
{
  size_t ixy = blockIdx.x*blockDim.x+threadIdx.x;

  //printf("gpu %d and %d \n", ixy, genome[ixy]);

  int mutpose= (int) (hiprand_uniform(&state[ixy+2])*(lengh));


  int muttype= (int) (hiprand_uniform(&state[ixy+2])*(3))+1;

  //float test=hiprand_uniform(&state[ixy]+ixy)*(lengh);

  genome[mutpose]=(genome[mutpose]+muttype)%4;

  printf("gpu %d\n", mutpose);
  //printf("gpu %f\n", test);
}

__global__ void grand(hiprandState *state,unsigned long seed)
{
    size_t ixy = blockIdx.x*blockDim.x+threadIdx.x;

    hiprand_init(seed,(ixy),0,&state[ixy+2]);
}


int main()
{
  int mutationnum;
  int lengh;
  int day;
  int *genome;

  int createmut=5;
  hiprandState *state;




  FILE* ifile = fopen ("genome.txt", "r");

  fscanf (ifile, "%d", &mutationnum);
  fscanf (ifile, "%d", &day);
  fscanf (ifile, "%d", &lengh);


  genome=(int*)malloc(lengh*sizeof(int));


    for(int i=0;i<lengh;i++)
    {
      fscanf (ifile, "%d", &genome[i]);
    }

    hipSetDevice(0);


    int *gpm;
    hipMalloc(&gpm,  lengh*sizeof(int));

    hipMalloc((void **)&state,(createmut+2)*sizeof(hiprandState));
    hipMemcpy(gpm,genome,lengh*sizeof(int),hipMemcpyHostToDevice);
    grand<<< createmut, 1 >>>(state,unsigned(time(NULL)));
    hipDeviceSynchronize();
    mutate<<< createmut, 1 >>>(gpm,lengh,state);
    hipMemcpy(genome,gpm,lengh*sizeof(int),hipMemcpyDeviceToHost);







    hipDeviceSynchronize();

    return 1;
}

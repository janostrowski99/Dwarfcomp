#include "hip/hip_runtime.h"
/**

nvcc -arch sm_50 -O3 mutation.cu -o mutation -lcuda -lcufft
*
* */

#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void mutate(int *genome,int lengh,hiprandState *state)
{
  int ixy = blockIdx.x*blockDim.x+threadIdx.x;

  //printf("gpu %d and %d \n", ixy, genome[ixy]);

  int mutpose= (int) (hiprand_uniform(&state[ixy+2])*(lengh));


  int muttype= (int) (hiprand_uniform(&state[ixy+2])*(3))+1;


  //float test=hiprand_uniform(&state[ixy]+ixy)*(lengh);

  genome[mutpose+(blockIdx.x*lengh)]=(genome[mutpose+(blockIdx.x*lengh)]+muttype)%4;

  //printf("gpu %d\n", mutpose);
  //printf("gpu %f\n", test);
}

__global__ void grand(hiprandState *state,unsigned long seed)
{
  int ixy = blockIdx.x*blockDim.x+threadIdx.x;

  hiprand_init(seed,(ixy),0,&state[ixy+2]);
}

__global__ void alcate(int *genome,int lengh)
{
  int ixy = (blockIdx.x+1)*(blockDim.x)+threadIdx.x;

  genome[ixy]=genome[threadIdx.x];


  //printf("a %d %d\n",blockIdx.x,  genome[2]);

}
__global__ void alcateBig(int *genome,int lengh)
{
  int ixy = (blockIdx.x+1)*lengh;
  int ixy2 = threadIdx.x;
  int ixy3;
  int big=lengh/1000;
  for(int b=0;b<big;b++)
  {
    ixy3=ixy2+b*1000;

    if(ixy3<lengh)
    {
      //printf("ixy %d ixy3 %d b %d gen %d \n",ixy, ixy3,b,genome[ixy3]);
      genome[ixy3+ixy]=genome[ixy3];
    }


  }




  //printf("a %d %d\n",blockIdx.x,  genome[2]);

}


__global__ void calculateEntropy(int *genome,int posi, int types, int len, int *entropy)
{

  extern  __shared__ int temp[];

  //__shared__ int temp[12];
  int ixy = blockIdx.x*blockDim.x+threadIdx.x;
  int total=0;
  int multi;
  //printf("total %d", blockIdx.x);
  for(int j=ixy;j<ixy+len;j++)
  {
    multi=1;
    for(int i=0;i<j-ixy;i++)
    {
      multi=multi*types;
    }
    //printf("multi %d", types);
    total=total+(genome[j]*multi); //entropy state numbers
    //temp[total+(blockIdx.x*posi)]+=1; //MMMMMMEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEHHHHHHHHHH

  }
  //printf("%d \n",blockDim.x);
  __threadfence();
  atomicAdd(temp+(total+(blockIdx.x*posi)),1);
  //temp[total+(blockIdx.x*posi)]+=1; //MMMMMMEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEHHHHHHHHHH

  //temp[1]++;
  entropy[total+(blockIdx.x*posi)]=temp[total+(blockIdx.x*posi)];
  //printf("total %d %d %d \n",genome[ixy], entropy[total+(blockIdx.x*posi)], total+(blockIdx.x*posi));
  //printf("total %d %d %d \n",genome[ixy], entropy[total+(blockIdx.x*posi)],  temp[1]);
  __syncthreads();
}

__global__ void calculateEntropyBig(int *genome,int posi, int types, int len, int lengh, int *entropy)
{

  extern  __shared__ int temp[];

  //__shared__ int temp[12];
  int ixy = blockIdx.x*lengh;
  int ixy2 = threadIdx.x;
  int ixy3;
  int total=0;
  int big=lengh/1000;
  //printf("big%d", big);
  int multi;


  for(int b=0;b<big;b++)
  {
    ixy3=ixy2+b*1000;

    if(ixy3<lengh)
    {
      total=0;


    for(int j=ixy+ixy3;j<ixy+ixy3+len;j++)
    {
      multi=1;
      for(int i=0;i<j-ixy-ixy3;i++)
      {
        multi=multi*types;
      }
      //printf("multi %d", types);
      total=total+(genome[j]*multi); //entropy state numbers

      //temp[total+(blockIdx.x*posi)]+=1; //MMMMMMEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEHHHHHHHHHH

    }
    //printf("ixy %d ixy3 %d b %d total %d \n",ixy, ixy3,b,total);
    atomicAdd(temp+(total+(blockIdx.x*posi)),1);
    entropy[total+(blockIdx.x*posi)]=temp[total+(blockIdx.x*posi)];
  }
  }
  __syncthreads();
}

__device__ double calculateEntropy2( int posi ,int* entab,int set) // possibilities and array for entropy
{
  double sum=0;
  int mv=set*posi;
  for(int i =0;i<posi;i++)
  {
    sum=sum+entab[i+mv];
  }

  double entropy=0;
  for(int i =0;i<posi;i++)
  {
    //printf("ent %d %d \n",set , entab[i+mv]);

    if((double)entab[i+mv]/sum>0)
    {
      entropy=entropy+(((double)entab[i+mv]/sum)*(double)log2((double)entab[i+mv]/sum));
    }

  }
  return -entropy;
}

__global__ void calculateEntropy3(int posi, int *entropy, double*out)
{
  int ixy = blockIdx.x*blockDim.x+threadIdx.x;

  //printf("%d %d\n",entropy[ixy],ixy);
  out[ixy]=calculateEntropy2(posi,entropy,ixy);
  //printf("entr %d %lf \n", ixy ,out[ixy]);

}




int main(int argc, char* argv[])
{
  int mutationnum;
  int lengh;
  int day;
  int *genome;
  hipError_t err;


  //chengables
  int createmut=5; //how many mutations to create
  int nchain=4; //how many genoms to mutate
  int len=1; //leng of chcked chain
  int types=4; //number of type variables
  //~chengables

  sscanf(argv[1], "%d", &createmut);
  sscanf(argv[2], "%d", &nchain);
  sscanf(argv[3], "%d", &len);


  int posi=types;
  for(int i=1;i<len;i++)
  {
    posi=posi*types;
  }


  hiprandState *state;




  FILE* ifile = fopen ("genome.txt", "r");

  fscanf (ifile, "%d", &mutationnum);
  fscanf (ifile, "%d", &day);
  fscanf (ifile, "%d", &lengh);
  //lengh=150;

  genome=(int*)malloc(lengh*sizeof(int));


  for(int i=0;i<lengh;i++)
  {
    fscanf (ifile, "%d", &genome[i]);
    //printf("%d %d\n",i, genome[i]);
  }



  //hipSetDevice(0);




  double *outentropy;
  int *entropytab;
  double enbuffer[posi][nchain];
  for(int i=0;i<posi;i++)
  {
    for(int j=0;j<nchain;j++)
    {
      enbuffer[i][j]=0;
    }
  }
  hipMalloc(&entropytab, nchain*posi*sizeof(int));
  hipMalloc(&outentropy, nchain*sizeof(double));
  hipMemcpy(enbuffer,entropytab,nchain*posi*sizeof(int),hipMemcpyHostToDevice);

  int *gpm;
  hipMalloc(&gpm,nchain*lengh*sizeof(int));
  err=hipMemcpy(gpm,genome,lengh*sizeof(int),hipMemcpyHostToDevice);

  if(err == hipErrorInvalidValue)
  printf("1!\n");
  else if(err == hipErrorInvalidDevicePointer)
  printf("2!\n");
  else if(err == hipErrorInvalidMemcpyDirection)
  printf("3!\n");

  hipMalloc((void **)&state,(createmut+2)*sizeof(hiprandState));
  hipDeviceSynchronize();

  if(lengh<1024)
  {
    alcate<<<  nchain-1,lengh >>>(gpm,lengh);
  }
  else
  {
    alcateBig<<<  nchain-1,1000 >>>(gpm,lengh);
  }


  hipDeviceSynchronize();
  grand<<< createmut, nchain >>>(state,unsigned(time(NULL)));
  hipDeviceSynchronize();
  mutate<<< createmut, nchain >>>(gpm,lengh,state);
  hipDeviceSynchronize();


  if(lengh<1024)
  {
    calculateEntropy<<<  nchain, lengh+1-len,posi*nchain*sizeof(int) >>>(gpm,posi,types,len,entropytab);
  }
  else
  {
    calculateEntropyBig<<<  nchain, 1000 ,posi*nchain*sizeof(int) >>>(gpm,posi,types,len,lengh,entropytab);
  }

  //calculateEntropy<<<  nchain, lengh+1-len,128 >>>(gpm,posi,types,len,entropytab);
  //calculateEntropy<<<  nchain, lengh+1-len >>>(gpm,posi,types,len,entropytab,temp);
  hipDeviceSynchronize();
  calculateEntropy3<<< 1, nchain >>>(posi,entropytab,outentropy);
  hipDeviceSynchronize();

  //hipMemcpy(genome,gpm,lengh*sizeof(int),hipMemcpyDeviceToHost);
  //hipMemcpy(enbuffer,entropy,posi*sizeof(double),hipMemcpyDeviceToHost);

  double outentropy2[nchain];

  hipMemcpy(outentropy2,outentropy,nchain*sizeof(double),hipMemcpyDeviceToHost);
  for(int i=0;i<nchain;i++)
  {
    printf("entr %lf\n", outentropy2[i]);
  }





  hipDeviceSynchronize();

  return 1;
}
